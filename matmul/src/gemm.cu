#include "hip/hip_runtime.h"
#include "gemm.cuh"

/*
    Matrix multiplies take two matrices of dimensions
    
    ixj * mxn => in
    
    and as long as j == m, the outputted matrix dims are ixn.
    
    ---

    Example 1:

    x (3 x 2)
    y (2 x 4)
    produces
    z (3 x 4)

    x1 x4 y1 y3 y5 y7
    x2 x5 y2 y4 y6 y8
    x3 x6
    
    z1 z4 z7 z10
    z2 z5 z8 z11
    z3 z6 z9 z12
    
    ==
    
    x1*y1 + x4*y2 | x1*y3 + x4*y4 | x1*y5 + x4*y6 | x1*y7 + x4*y8
    x2*y1 + x5*y2 | x2*y3 + x5*y4 | x2*y5 + x5*y6 | x2*y7 + x5*y8
    x3*y1 + x6*y2 | x3*y3 + x6*y4 | x3*y5 + x6*y6 | x3*y7 + x6*y8
    
    for each row in x, do dot product of each column in y1 (kind of)
*/

/*
    Raw, first intuition of GEMM on CPU.
*/
template <typename T>
__global__ void cpuGEMM(const T *X, const size_t x_nrows, const size_t x_ncols,
                        const T *Y, const size_t y_nrows, const size_t y_ncols,
                        T *Z, const size_t z_nrows, const size_t z_ncols) {
    for (size_t rx = 0; rx < x_nrows; rx++) {
        for (size_t cx = 0; cx < x_ncols; cx++) {
            size_t idx = cx * x_nrows + rx;

            
        }
    }
}

template <typename T>
__global__ void kernelGEMMv0(const T *X, const size_t x_nrows, const size_t x_ncols,
                        const T *Y, const size_t y_nrows, const size_t y_ncols,
                        T *Z, const size_t z_nrows, const size_t z_ncols) {
    // Which row 
    int r = blockIdx.x * blockDim.x + threadIdx.x;
    int c = blockIdx.y * blockDim.y + threadIdx.y;
    
}