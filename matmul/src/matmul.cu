#include <iostream>
#include <hip/hip_runtime.h>
#include <Eigen/Dense>

using std::cout, std::endl;

// vary as needed
using scalar_t = float;
int main() {
    // Eigen to make human rebaedable
    // Test case in gemm.cuh
    Eigen::Matrix<scalar_t, 3, 2> X = Eigen::Matrix<scalar_t, 3, 2>::Zero();
    Eigen::Matrix<scalar_t, 2, 4> Y = Eigen::Matrix<scalar_t, 2, 4>::Zero();
    Eigen::Matrix<scalar_t, 3, 4> Z = Eigen::Matrix<scalar_t, 3, 4>::Zero();

    return 0;
}